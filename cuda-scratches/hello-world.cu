
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloWorldFromGPU(void)
{
    printf("Running on GPU: threadId (%d,%d) - blockId (%d,%d) - blockDim (%d,%d)\n",
        threadIdx.x, threadIdx.y,
        blockIdx.x, blockIdx.y,
        blockDim.x, blockDim.y
    );
}

int main(void)
{
    printf("Hello World from CPU!\n");
    helloWorldFromGPU<<<5, 10>>>();
    hipDeviceSynchronize();
    return 0;
}